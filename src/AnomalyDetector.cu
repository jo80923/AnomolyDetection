#include "hip/hip_runtime.h"
#include "AnomalyDetector.cuh"

template<typename T>
__global__ void slideWindowDetector(unsigned int shortTermWindow, unsigned int longTermWindow, unsigned int numElements, T* data, double* anomalies, double2* moments, double2* flkclk, unsigned int order){
  int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
  int globalId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
  (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
  if(globalId < numElements){
    unsigned int ls = shortTermWindow;
    unsigned int ll = longTermWindow;

    double2 mean={0.0,0.0};
    T* reg_data = new T[ll];
    for(int i = globalId; i < ll + globalId; ++i){
      reg_data[i - globalId] = data[i];
      if(ls + globalId > i){
        mean.x += reg_data[i - globalId];
      }
      mean.y += reg_data[i - globalId];
    }
    mean.x /= (double)ls;
    mean.y /= (double)ll;

    double stk = 0.0;
    double ltk = 0.0;
    double2 dev= {0.0,0.0};
    double temp = 0.0;
    double currentValue = 0.0;
    for(int i = 0; i < ll; ++i){
      currentValue = reg_data[i];
      if(ls > i){
        temp = (currentValue - mean.x);
        dev.x += temp*temp;
        stk += temp*temp*temp*temp;
      }
      temp = (currentValue - mean.y);
      dev.y += temp*temp;
      ltk += temp*temp*temp*temp;
    }
    dev.x /= ((double)ls-1);
    dev.y /= ((double)ll-1);
    stk /= (((double)ls - 1)*dev.x*dev.x);
    ltk /= (((double)ll - 1)*dev.y*dev.y);

    double ratio = stk/(ltk + 0.0000000000001);

    dev.x = sqrt(dev.x);
    dev.y = sqrt(dev.y);

    unsigned int reg_order = order;
    double2 rawMoments[5] = {{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0}};
    for(int o = 0; o < reg_order; ++o){
      for(int i = 0; i < ll; ++i){
        if(ls > i){
          rawMoments[o].x += pow(reg_data[i],o+1)*(1.0/dev.x*sqrt(2*M_PI))*exp(-1*(reg_data[i] - mean.x)*(reg_data[i] - mean.x)/(2*dev.x*dev.x));
        }
        rawMoments[o].y += pow(reg_data[i],o+1)*(1.0/dev.y*sqrt(2*M_PI))*exp(-1*(reg_data[i] - mean.y)*(reg_data[i] - mean.y)/(2*dev.y*dev.y));
      }
    }

    double2 moment = {0.0,0.0};
    if(reg_order == 1) moment = rawMoments[0];
    else if(reg_order == 2){
      moment = {rawMoments[1].x - (rawMoments[0].x*rawMoments[0].x),rawMoments[1].y - (rawMoments[0].y*rawMoments[0].y)};
    }
    else if(reg_order == 3){
      moment = {rawMoments[2].x - (3*rawMoments[0].x*rawMoments[1].x) +
      (2*rawMoments[0].x*rawMoments[0].x*rawMoments[0].x),
      rawMoments[2].y - (3*rawMoments[0].y*rawMoments[1].y) +
      (2*rawMoments[0].y*rawMoments[0].y*rawMoments[0].y)};
    }
    else if(reg_order == 4){
      moment = {rawMoments[3].x - (4*rawMoments[0].x*rawMoments[2].x) +
      (6*rawMoments[0].x*rawMoments[0].x*rawMoments[1].x) -
      (3*rawMoments[0].x*rawMoments[0].x*rawMoments[0].x*rawMoments[0].x),
      rawMoments[3].y - (4*rawMoments[0].y*rawMoments[2].y) +
      (6*rawMoments[0].y*rawMoments[0].y*rawMoments[1].y) -
      (3*rawMoments[0].y*rawMoments[0].y*rawMoments[0].y*rawMoments[0].y)};
    }
    else if(reg_order == 5){
      moment = {rawMoments[4].x - (5*rawMoments[0].x*rawMoments[3].x) +
      (10*rawMoments[0].x*rawMoments[0].x*rawMoments[2].x) -
      (10*rawMoments[0].x*rawMoments[0].x*rawMoments[0].x*rawMoments[1].x),
      rawMoments[4].y - (5*rawMoments[0].y*rawMoments[3].y) +
      (10*rawMoments[0].y*rawMoments[0].y*rawMoments[2].y) -
      (10*rawMoments[0].y*rawMoments[0].y*rawMoments[0].y*rawMoments[1].y)};
    }
    else{
      delete[] reg_data;
      printf("bad order\n");
      asm("trap;");
    }
    flkclk[globalId] = {stk,ltk};
    anomalies[globalId] = ratio;
    moments[globalId] = moment;
    delete[] reg_data;
  }
}


jaxdsp::SlidingWindow::SlidingWindow(){
  this->shortTermWindow = 1;
  this->longTermWindow = 2;
  this->order = 1;
}

jaxdsp::SlidingWindow::~SlidingWindow(){

}

void jaxdsp::SlidingWindow::setShortTerm(const unsigned int &shortTermWindow){
  this->shortTermWindow = shortTermWindow;
}
void jaxdsp::SlidingWindow::setLongTerm(const unsigned int &longTermWindow){
  this->longTermWindow = longTermWindow;
}
void jaxdsp::SlidingWindow::setOrder(const unsigned int &order){
  this->order = order;
}

void jaxdsp::SlidingWindow::detectAnomaly(jax::Unity<double>* data, jax::Unity<double2>* &moments, jax::Unity<double>* &anomalies, jax::Unity<double2>* &kurtValues){
  assert(data != NULL);
  data->transferMemoryTo(jax::gpu);

  double* anomalies_host = new double[data->numElements - this->longTermWindow + 1]();
  anomalies = new jax::Unity<double>(anomalies_host,data->numElements - this->longTermWindow + 1,jax::cpu);
  double2* moments_host = new double2[data->numElements - this->longTermWindow + 1]();
  moments = new jax::Unity<double2>(moments_host,data->numElements - this->longTermWindow + 1,jax::cpu);
  double2* kurts_host = new double2[data->numElements - this->longTermWindow + 1]();
  kurtValues = new jax::Unity<double2>(kurts_host, data->numElements - this->longTermWindow + 1, jax::cpu);

  kurtValues->transferMemoryTo(jax::gpu);
  anomalies->transferMemoryTo(jax::gpu);
  moments->transferMemoryTo(jax::gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(data->numElements - this->longTermWindow + 1, grid, block);

  slideWindowDetector<<<grid, block>>>(this->shortTermWindow, this->longTermWindow, data->numElements - this->longTermWindow + 1, data->device, anomalies->device, moments->device, kurtValues->device, this->order);
  kurtValues->transferMemoryTo(jax::cpu);
  anomalies->transferMemoryTo(jax::cpu);
  moments->transferMemoryTo(jax::cpu);
}
